#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024
#define THREADS_PER_BLOCK 512

__global__ void reduce_sum(int *data) {
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    __shared__ int sdata[THREADS_PER_BLOCK];

    sdata[tid] = data[i];
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        data[blockIdx.x] = sdata[0];
    }
}

int main() {
    int size = N * sizeof(int);
    int *h_data = (int *)malloc(size);

    for (int i = 0; i < N; i++) {
        h_data[i] = 1;
    }

    int *d_data;
    hipMalloc((void **)&d_data, size);
    hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);

    int blocks = N / THREADS_PER_BLOCK;
    reduce_sum<<<blocks, THREADS_PER_BLOCK>>>(d_data);

    if (blocks > 1) {
        reduce_sum<<<1, blocks>>>(d_data);
    }

    int result;
    hipMemcpy(&result, d_data, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum = %d\n", result);

    hipFree(d_data);
    free(h_data);
    return 0;
}
